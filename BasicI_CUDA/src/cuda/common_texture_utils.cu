#include "hip/hip_runtime.h"
#include "common_texture_utils.h"

namespace surfelwarp
{

	// shared pointer version of CudaTextureSurface
	CudaTextureSurfaceT::~CudaTextureSurfaceT()
	{
		release();
	}

	void CudaTextureSurfaceT::release()
	{
		//printf("[CudaTextureSurfaceT] release %d (%d %d %d)\n", (int)this,
		//	(int)this->texture, (int)this->surface, (int)this->d_array);

		if (this->texture)
			cudaSafeCall(hipDestroyTextureObject(this->texture));
		if (this->surface)
			cudaSafeCall(hipDestroySurfaceObject(this->surface));
		if (this->d_array)
			cudaSafeCall(hipFreeArray(this->d_array));

		this->texture = 0;
		this->surface = 0;
		this->d_array = 0;
	}

	hipTextureObject_t create1DLinearTexture(const DeviceArray<float> &array)
	{
		hipTextureDesc texture_desc;
		memset(&texture_desc, 0, sizeof(hipTextureDesc));
		texture_desc.normalizedCoords = 0;
		texture_desc.addressMode[0] = hipAddressModeBorder; //Return 0 outside the boundary
		texture_desc.addressMode[1] = hipAddressModeBorder;
		texture_desc.addressMode[2] = hipAddressModeBorder;
		texture_desc.filterMode = hipFilterModePoint;
		texture_desc.readMode = hipReadModeElementType;
		texture_desc.sRGB = 0;

		//Create resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeLinear;
		resource_desc.res.linear.devPtr = (void*)array.ptr();
		resource_desc.res.linear.sizeInBytes = array.sizeBytes();
		resource_desc.res.linear.desc.f = hipChannelFormatKindFloat;
		resource_desc.res.linear.desc.x = 32;
		resource_desc.res.linear.desc.y = 0;
		resource_desc.res.linear.desc.z = 0;
		resource_desc.res.linear.desc.w = 0;

		//Allocate the texture
		hipTextureObject_t d_texture;
		cudaSafeCall(hipCreateTextureObject(&d_texture, &resource_desc, &texture_desc, nullptr));
		return d_texture;
	}

	hipTextureObject_t create1DLinearTexture(const DeviceBufferArray<float>& array)
	{
		DeviceArray<float> pcl_array((float*)array.Ptr(), array.Capacity());
		return create1DLinearTexture(pcl_array);
	}

	void createDefault2DTextureDesc(hipTextureDesc &desc, hipTextureAddressMode border_mode)
	{
		// hipAddressModeBorder: Return 0 outside the boundary

		memset(&desc, 0, sizeof(desc));
		desc.addressMode[0] = border_mode;
		desc.addressMode[1] = border_mode;
		desc.addressMode[2] = border_mode;
		desc.filterMode = hipFilterModePoint;
		desc.readMode = hipReadModeElementType;
		desc.normalizedCoords = 0;
	}

	void createLinear2DTextureDesc(hipTextureDesc &desc, hipTextureAddressMode border_mode)
	{
		memset(&desc, 0, sizeof(desc));
		desc.addressMode[0] = border_mode;
		desc.addressMode[1] = border_mode;
		desc.addressMode[2] = border_mode;
		desc.filterMode = hipFilterModeLinear;
		desc.readMode = hipReadModeElementType;
		desc.normalizedCoords = 0;
	}

	////////////////////////////// bind collect //////////////////////////////
	void bind2DTextureSurface(hipTextureObject_t &texture, hipSurfaceObject_t &surface,
		hipArray_t &d_array, hipTextureAddressMode border_mode)
	{
		//The texture description
		hipTextureDesc depth_texture_desc;
		createDefault2DTextureDesc(depth_texture_desc, border_mode);

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}

	void bind2DTextureSurface(CudaTextureSurface & collect, hipTextureAddressMode border_mode)
	{
		bind2DTextureSurface(collect.texture, collect.surface, collect.d_array, border_mode);
	}

	void bind2DTextureSurface(std::shared_ptr<CudaTextureSurfaceT>& collect, hipTextureAddressMode border_mode)
	{
		if (!collect)
			return;
		bind2DTextureSurface(collect->texture, collect->surface, collect->d_array, border_mode);
	}

	////////////////////////////// depth ushort1 //////////////////////////////
	void createUShort1Texture(const unsigned height, const unsigned width,
		hipTextureObject_t &texture, hipArray_t &d_array)
	{
		//The texture description
		hipTextureDesc depth_texture_desc;
		createDefault2DTextureDesc(depth_texture_desc);

		//Create channel descriptions
		hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

		//Allocate the cuda array
		cudaSafeCall(hipMallocArray(&d_array, &depth_channel_desc, width, height));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_desc, 0));
	}

	void createUShort1TextureSurface(const unsigned height, const unsigned width,
		hipTextureObject_t &texture,
		hipSurfaceObject_t &surface,
		hipArray_t &d_array) 
	{
		//The texture description
		hipTextureDesc depth_texture_desc;
		createDefault2DTextureDesc(depth_texture_desc);

		//Create channel descriptions
		hipChannelFormatDesc depth_channel_desc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

		//Allocate the cuda array
		cudaSafeCall(hipMallocArray(&d_array, &depth_channel_desc, width, height));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &depth_texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}

	void createUShort1TextureSurface(const unsigned height, const unsigned width, CudaTextureSurface & texture_collect)
	{
		createUShort1TextureSurface(height, width,
			texture_collect.texture, texture_collect.surface, texture_collect.d_array);
	}

	void createUShort1TextureSurface(const unsigned height, const unsigned width,
		std::shared_ptr<CudaTextureSurfaceT>& texture_collect)
	{
		if (!texture_collect)
			texture_collect = std::make_shared<CudaTextureSurfaceT>();

		if (height == 0 || width == 0) {
			texture_collect->release();
			return;
		}

		createUShort1TextureSurface(
			height, width,
			texture_collect->texture,
			texture_collect->surface,
			texture_collect->d_array
		);
	}

	////////////////////////////// float4 texture //////////////////////////////
	void createFloat4TextureSurface(const unsigned height, const unsigned width,
		hipTextureObject_t &texture,
		hipSurfaceObject_t &surface,
		hipArray_t &d_array) 
	{
		//The texture description
		hipTextureDesc float4_texture_desc;
		createDefault2DTextureDesc(float4_texture_desc, hipAddressModeClamp);

		//Create channel descriptions
		hipChannelFormatDesc float4_channel_desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

		//Allocate the cuda array
		cudaSafeCall(hipMallocArray(&d_array, &float4_channel_desc, width, height));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float4_texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}

	void createFloat4TextureSurface(const unsigned rows, const unsigned cols, CudaTextureSurface & texture_collect)
	{
		createFloat4TextureSurface(
			rows, cols,
			texture_collect.texture,
			texture_collect.surface,
			texture_collect.d_array
		);
	}

	void createFloat4TextureSurface(const unsigned height, const unsigned width,
		std::shared_ptr<CudaTextureSurfaceT>& texture_collect) 
	{
		if (!texture_collect)
			texture_collect = std::make_shared<CudaTextureSurfaceT>();

		if (height == 0 || width == 0) {
			texture_collect->release();
			return;
		}

		createFloat4TextureSurface(
			height, width,
			texture_collect->texture,
			texture_collect->surface,
			texture_collect->d_array
		);
	}

	////////////////////////////// float1 //////////////////////////////
	void createFloat1TextureSurface(const unsigned height, const unsigned width,
		hipTextureObject_t& texture, hipSurfaceObject_t& surface,
		hipArray_t& d_array, bool linear)
	{
		//The texture description
		hipTextureDesc float1_texture_desc;
		if (!linear)
			createDefault2DTextureDesc(float1_texture_desc);
		else
			createLinear2DTextureDesc(float1_texture_desc);

		//Create channel descriptions
		hipChannelFormatDesc float1_channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

		//Allocate the cuda array
		cudaSafeCall(hipMallocArray(&d_array, &float1_channel_desc, width, height));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float1_texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}


	void createFloat1TextureSurface(const unsigned height, const unsigned width,
		CudaTextureSurface & texture_collect, bool linear)
	{
		createFloat1TextureSurface(
			height, width,
			texture_collect.texture,
			texture_collect.surface,
			texture_collect.d_array,
			linear
		);
	}

	void createFloat1TextureSurface(const unsigned height, const unsigned width,
		std::shared_ptr<CudaTextureSurfaceT>& texture_collect, bool linear)
	{
		if (!texture_collect)
			texture_collect = std::make_shared<CudaTextureSurfaceT>();

		if (height == 0 || width == 0) {
			texture_collect->release();
			return;
		}

		createFloat1TextureSurface(
			height, width,
			texture_collect->texture,
			texture_collect->surface,
			texture_collect->d_array,
			linear
		);
	}

	////////////////////////////// float2 //////////////////////////////
	void createFloat2TextureSurface(const unsigned height, const unsigned width,
		hipTextureObject_t & texture,
		hipSurfaceObject_t & surface,
		hipArray_t & d_array) 
	{
		//The texture description
		hipTextureDesc float2_texture_desc;
		createDefault2DTextureDesc(float2_texture_desc);

		//Create channel descriptions
		hipChannelFormatDesc float2_channel_desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);

		//Allocate the cuda array
		cudaSafeCall(hipMallocArray(&d_array, &float2_channel_desc, width, height));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float2_texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}

	void createFloat2TextureSurface(const unsigned height, const unsigned width,
		CudaTextureSurface & texture_collect)
	{
		createFloat2TextureSurface(
			height, width,
			texture_collect.texture,
			texture_collect.surface,
			texture_collect.d_array
		);
	}

	void createFloat2TextureSurface(const unsigned height, const unsigned width,
		std::shared_ptr<CudaTextureSurfaceT>& texture_collect)
	{
		if (!texture_collect)
			texture_collect = std::make_shared<CudaTextureSurfaceT>();

		if (height == 0 || width == 0) {
			texture_collect->release();
			return;
		}

		createFloat2TextureSurface(
			height, width,
			texture_collect->texture,
			texture_collect->surface,
			texture_collect->d_array
		);
	}


	void createUChar1TextureSurface(const unsigned height, const unsigned width,
		hipTextureObject_t & texture,
		hipSurfaceObject_t & surface,
		hipArray_t & d_array)
	{
		//The texture description
		hipTextureDesc uchar1_texture_desc;
		createDefault2DTextureDesc(uchar1_texture_desc, hipAddressModeClamp);

		//Create channel descriptions
		hipChannelFormatDesc uchar1_channel_desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

		//Allocate the cuda array
		cudaSafeCall(hipMallocArray(&d_array, &uchar1_channel_desc, width, height));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &uchar1_texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}

	void createUChar1TextureSurface(const unsigned height, const unsigned width,
		CudaTextureSurface & texture_collect)
	{
		createUChar1TextureSurface(
			height, width,
			texture_collect.texture,
			texture_collect.surface,
			texture_collect.d_array
		);
	}

	void createUChar1TextureSurface(const unsigned height, const unsigned width,
		std::shared_ptr<CudaTextureSurfaceT>& texture_collect)
	{
		if (!texture_collect)
			texture_collect = std::make_shared<CudaTextureSurfaceT>();

		if (height == 0 || width == 0) {
			texture_collect->release();
			return;
		}

		createUChar1TextureSurface(
			height, width,
			texture_collect->texture,
			texture_collect->surface,
			texture_collect->d_array
		);
	}


	void createUChar4TextureSurface(const unsigned height, const unsigned width,
		hipTextureObject_t & texture,
		hipSurfaceObject_t & surface,
		hipArray_t & d_array)
	{
		//The texture description
		hipTextureDesc uchar4_texture_desc;
		createDefault2DTextureDesc(uchar4_texture_desc, hipAddressModeClamp);
		//createLinear2DTextureDesc(uchar4_texture_desc, hipAddressModeClamp);

		//Create channel descriptions
		hipChannelFormatDesc uchar4_channel_desc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

		//Allocate the cuda array
		cudaSafeCall(hipMallocArray(&d_array, &uchar4_channel_desc, width, height));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &uchar4_texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}

	void createUChar4TextureSurface(
		const unsigned height, const unsigned width,
		CudaTextureSurface & texture_collect)
	{
		createUChar4TextureSurface(
			height, width,
			texture_collect.texture,
			texture_collect.surface,
			texture_collect.d_array
		);
	}

	void createUChar4TextureSurface(const unsigned height, const unsigned width,
		std::shared_ptr<CudaTextureSurfaceT>& texture_collect)
	{
		if (!texture_collect)
			texture_collect = std::make_shared<CudaTextureSurfaceT>();

		if (height == 0 || width == 0) {
			texture_collect->release();
			return;
		}
		cudaSafeCall(hipDeviceSynchronize());
		cudaSafeCall(hipGetLastError());
		createUChar4TextureSurface(
			height, width,
			texture_collect->texture,
			texture_collect->surface,
			texture_collect->d_array
		);
		cudaSafeCall(hipDeviceSynchronize());
		cudaSafeCall(hipGetLastError());
	}

	////////////////////////////// int1 //////////////////////////////
	void createInt1TextureSurface(const unsigned height, const unsigned width,
		hipTextureObject_t& texture, hipSurfaceObject_t& surface,
		hipArray_t& d_array, bool linear)
	{
		//The texture description
		hipTextureDesc texture_desc;
		if (!linear)
			createDefault2DTextureDesc(texture_desc);
		else
			createLinear2DTextureDesc(texture_desc);

		//Create channel descriptions
		hipChannelFormatDesc channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);

		//Allocate the cuda array
		cudaSafeCall(hipMallocArray(&d_array, &channel_desc, width, height));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}


	void createInt1TextureSurface(const unsigned height, const unsigned width,
		CudaTextureSurface& texture_collect, bool linear)
	{
		createInt1TextureSurface(
			height, width,
			texture_collect.texture,
			texture_collect.surface,
			texture_collect.d_array,
			linear
		);
	}

	void createInt1TextureSurface(const unsigned height, const unsigned width,
		std::shared_ptr<CudaTextureSurfaceT>& texture_collect, bool linear)
	{
		if (!texture_collect)
			texture_collect = std::make_shared<CudaTextureSurfaceT>();

		if (height == 0 || width == 0) {
			texture_collect->release();
			return;
		}

		createInt1TextureSurface(
			height, width,
			texture_collect->texture,
			texture_collect->surface,
			texture_collect->d_array,
			linear
		);
	}

	////////////////////////////// query //////////////////////////////
	void query2DTextureExtent(hipTextureObject_t texture, unsigned &width, unsigned &height)
	{
		hipResourceDesc texture_res;
		cudaSafeCall(hipGetTextureObjectResourceDesc(&texture_res, texture));
		hipArray_t cu_array = texture_res.res.array.array;
		hipChannelFormatDesc channel_desc;
		hipExtent extent;
		unsigned int flag;
		cudaSafeCall(hipArrayGetInfo(&channel_desc, &extent, &flag, cu_array));

		width = extent.width;
		height = extent.height;
	}

	void queryCudaArrayExtent(hipArray_t cu_array, unsigned& width, unsigned& height)
	{
		hipChannelFormatDesc channel_desc;
		hipExtent extent;
		unsigned int flag;
		cudaSafeCall(hipArrayGetInfo(&channel_desc, &extent, &flag, cu_array));

		width = extent.width;
		height = extent.height;
	}

	// release
	void releaseTextureCollect(CudaTextureSurface & texture_collect)
	{
		if (texture_collect.texture)
			cudaSafeCall(hipDestroyTextureObject(texture_collect.texture));
		if (texture_collect.surface)
			cudaSafeCall(hipDestroySurfaceObject(texture_collect.surface));
		if (texture_collect.d_array)
			cudaSafeCall(hipFreeArray(texture_collect.d_array));

		texture_collect.texture = 0;
		texture_collect.surface = 0;
		texture_collect.d_array = 0;
	}

	///////////////////////////////////// 3D /////////////////////////////////////
	/**
	* \brief Create TextureDesc for default 3D texture
	*/
	void createDefault3DTextureDesc(hipTextureDesc &desc)
	{
		memset(&desc, 0, sizeof(desc));
		desc.addressMode[0] = hipAddressModeBorder; //Return 0 outside the boundary
		desc.addressMode[1] = hipAddressModeBorder;
		desc.addressMode[2] = hipAddressModeBorder;
		desc.filterMode = hipFilterModePoint;
		desc.readMode = hipReadModeElementType;
		desc.normalizedCoords = 0;
	}

	/**
	* \brief Create 3D float1 textures (and surfaces) for mean-field inference
	*/
	void createFloat1TextureSurface3D(
		const unsigned width, const unsigned height, const unsigned depth,
		hipTextureObject_t& texture, hipSurfaceObject_t& surface,
		hipArray_t& d_array)
	{
		//The texture description
		hipTextureDesc float1_texture_desc;
		createDefault3DTextureDesc(float1_texture_desc);

		//Create channel descriptions
		hipChannelFormatDesc float1_channel_desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

		//Allocate the cuda array
		hipExtent ext;
		ext.width = (unsigned)width;
		ext.height = (unsigned)height;
		ext.depth = (unsigned)depth;
		cudaSafeCall(hipMalloc3DArray(&d_array, &float1_channel_desc, ext));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &float1_texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}


	void createFloat1TextureSurface3D(
		const unsigned width, const unsigned height, const unsigned depth,
		std::shared_ptr<CudaTextureSurfaceT>& texture_collect)
	{
		if (!texture_collect)
			texture_collect = std::make_shared<CudaTextureSurfaceT>();

		if (height == 0 || width == 0) {
			texture_collect->release();
			return;
		}

		createFloat1TextureSurface3D(
			width, height, depth,
			texture_collect->texture,
			texture_collect->surface,
			texture_collect->d_array
		);
	}

	/**
	* \brief Create 3D float1 textures (and surfaces) for mean-field inference
	*/
	void createUChar1TextureSurface3D(
		const unsigned width, const unsigned height, const unsigned depth,
		hipTextureObject_t& texture, hipSurfaceObject_t& surface,
		hipArray_t& d_array)
	{
		//The texture description
		hipTextureDesc uchar1_texture_desc;
		createDefault3DTextureDesc(uchar1_texture_desc);

		//Create channel descriptions
		hipChannelFormatDesc uchar1_channel_desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

		//Allocate the cuda array
		hipExtent ext;
		ext.width = (unsigned)width;
		ext.height = (unsigned)height;
		ext.depth = (unsigned)depth;
		cudaSafeCall(hipMalloc3DArray(&d_array, &uchar1_channel_desc, ext));

		//Create the resource desc
		hipResourceDesc resource_desc;
		memset(&resource_desc, 0, sizeof(hipResourceDesc));
		resource_desc.resType = hipResourceTypeArray;
		resource_desc.res.array.array = d_array;

		//Allocate the texture
		cudaSafeCall(hipCreateTextureObject(&texture, &resource_desc, &uchar1_texture_desc, 0));
		cudaSafeCall(hipCreateSurfaceObject(&surface, &resource_desc));
	}

	void createUChar1TextureSurface3D(
		const unsigned width, const unsigned height, const unsigned depth,
		std::shared_ptr<CudaTextureSurfaceT>& texture_collect)
	{
		if (!texture_collect)
			texture_collect = std::make_shared<CudaTextureSurfaceT>();

		if (height == 0 || width == 0) {
			texture_collect->release();
			return;
		}

		createUChar1TextureSurface3D(
			width, height, depth,
			texture_collect->texture,
			texture_collect->surface,
			texture_collect->d_array
		);
	}

}
